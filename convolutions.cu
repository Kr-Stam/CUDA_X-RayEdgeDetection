#include "hip/hip_runtime.h"
#include "convolutions.cuh"
#include "utils.hpp"

#include <hip/hip_runtime.h>
#include <>

///constant memory used for mask convolution
__constant__ double c_mask[25];

__constant__ double GAUS_KERNEL_3x3_d[9] = {
	0.0625, 0.125, 0.0625,
	0.125, 0.25, 0.125,
	0.0625, 0.125, 0.0625
};


//----------------CONVOLUTIONS--------------------

//-------------------3CHANNEL---------------------
/**
* \brief Unoptimized CUDA kernel for 2D convolution
*
* \param src Source Matrix
* \param mask Mask Matrix
* \param dest Destination Matrix
* \param w Width
* \param h Heigth
* \param mw Mask Width
* \param mh Mask Height
*/
__global__ void g_conv_3ch_2d(const unsigned char *src, const double *mask, unsigned char *dest, int w, int h, int mw, int mh)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= w || y >= h)
	{
			return;
	}

	int pos = y * w + x;

	int tmp[3] = {0, 0, 0};

	int hmw = mw >> 1;
	int hmh = mh >> 1;
	int start_x = x - hmw;
	int start_y = y - hmh;
	int tmp_pos, mask_pos, tmp_x, tmp_y;

	for (int i = 0; i < mh; i++)
	{
			for (int j = 0; j < mw; j++)
			{
					tmp_x = start_x + j;
					tmp_y = start_y + i;
					if (tmp_x >= 0 && tmp_x < w && tmp_y >= 0 && tmp_y < h)
					{
							tmp_pos = tmp_y * w + tmp_x;
							mask_pos = i * mw + j;
							tmp[0] += src[tmp_pos * 3] * mask[mask_pos];
							tmp[1] += src[tmp_pos * 3 + 1] * mask[mask_pos];
							tmp[2] += src[tmp_pos * 3 + 2] * mask[mask_pos];
					}
			}
	}
	dest[pos * 3] = (unsigned char)tmp[0];
	dest[pos * 3 + 1] = (unsigned char)tmp[1];
	dest[pos * 3 + 2] = (unsigned char)tmp[2];
}

/**
* \brief Launch a CUDA kernel to perform 2D convolution
*
* \param src Source Matrix
* \param dest Destination Matrix
* \param w Width
* \param h Height
* \param mask_t Mask Matrix
* \param mw Mask Width <=5
* \param mh Mask Height <=5
*/
void gpu::conv_3ch_2d(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const double *mask_t, int mw, int mh)
{

	size_t size = w * h * 3 * sizeof(unsigned char);

	unsigned char *src_d;
	unsigned char *dest_d;
	double *mask_d;

	hipMalloc((void **)&src_d, size);
	hipMalloc((void **)&dest_d, size);
	hipMalloc((void **)&mask_d, mw * mh * sizeof(double));

	hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask_t, mw * mh * sizeof(double), hipMemcpyHostToDevice);

	int NUM_OF_THREADS = 32;
	dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
	int GRID_SIZE_X = (int)ceil((float)w / NUM_OF_THREADS);
	int GRID_SIZE_Y = (int)ceil((float)h / NUM_OF_THREADS);
	dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
	g_conv_3ch_2d<<<blockSize, gridSize>>>(src_d, mask_d, dest_d, w, h, mw, mh);

	hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(dest_d);
	hipFree(mask_d);
}

/**
* \brief A more optimized 2D convolution where the mask is loaded into constant GPU memory before execution
*
* \param src Source Matrix
* \param dest Destination Matrix
* \param w Width
* \param h Height
* \param mw Mask Width
* \param mh Mask Height
*/ 
__global__ void g_conv_3ch_2d_constant(const unsigned char *src, unsigned char *dest, int w, int h, int mw, int mh)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= w || y >= h)
	{
			return;
	}

	int pos = y * w + x;

	int tmp[3] = {0, 0, 0};

	int hmw = mw >> 1;
	int hmh = mh >> 1;
	int start_x = x - hmw;
	int start_y = y - hmh;
	int tmp_pos, mask_pos, tmp_x, tmp_y;

	for (int i = 0; i < mh; i++)
	{
			for (int j = 0; j < mw; j++)
			{
					tmp_x = start_x + j;
					tmp_y = start_y + i;
					if (tmp_x >= 0 && tmp_x < w && tmp_y >= 0 && tmp_y < h)
					{
							tmp_pos = tmp_y * w + tmp_x;
							mask_pos = i * mw + j;
							tmp[0] += src[tmp_pos * 3] * c_mask[mask_pos];
							tmp[1] += src[tmp_pos * 3 + 1] * c_mask[mask_pos];
							tmp[2] += src[tmp_pos * 3 + 2] * c_mask[mask_pos];
					}
			}
	}
	dest[pos * 3] = (unsigned char)tmp[0];
	dest[pos * 3 + 1] = (unsigned char)tmp[1];
	dest[pos * 3 + 2] = (unsigned char)tmp[2];
}

/**
* \brief Launch a CUDA kernel to perform a 2D convolution with constant memory
*
* \param src Source Matrix
* \param dest Destination Matrix
* \param w Width
* \param h Height
* \param mask_t Mask Matrix
* \param mw Mask Width <=5
* \param mh Mask Height <=5
*/
void gpu::conv_3ch_2d_constant(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const double *mask_t, int mw, int mh)
{
	size_t size = w * h * 3 * sizeof(unsigned char);

	unsigned char *src_d;
	unsigned char *dest_d;

	hipMalloc((void **)&src_d, size);
	hipMalloc((void **)&dest_d, size);

	hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mask), mask_t, mw * mh * sizeof(double));

	int NUM_OF_THREADS = 32;
	dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
	int GRID_SIZE_X = (int)ceil((float)w / NUM_OF_THREADS);
	int GRID_SIZE_Y = (int)ceil((float)h / NUM_OF_THREADS);
	dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
	g_conv_3ch_2d_constant<<<blockSize, gridSize>>>(src_d, dest_d, w, h, mw, mh);

	hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(dest_d);
}

__global__ void g_conv_3ch_tiled(const unsigned char *src, unsigned char *dest, int w, int h, int mw, int mh, int TILE_SIZE_X, int TILE_SIZE_Y){
	//load all data
	//Objasnuvanje za kako raboti, povekje e ova za licna upotreba
	//Se upotrebuva maksimalniot mozhen blockSize shto e 32x32
	//Se loadiraat site vrednosti vnatre vo toj blockSize
	//Se koristi TILE_SIZE shto e 32-mw+1;
	//Za da se loadiraat vrednosti nadvor od src mora da se napravat input indeksi i output indeksi
	//Mapiranjeto na nivo na thread e out(0,0) e na TILE_SIZE, in(0,0) e na BLOCK_SIZE
	//Site threads loadiraat, ama ako threadot e nadvor od TILE_SIZE togash ne e output thread 

	extern __shared__ unsigned char tile[];    

	int hmh = mh >> 1;
	int hmw = mw >> 1;

	int x_o = threadIdx.x + blockIdx.x * TILE_SIZE_X;
	int y_o = threadIdx.y + blockIdx.y * TILE_SIZE_Y;
	int pos_o = x_o + y_o * w; 
	int x_i = x_o - hmw;
	int y_i = y_o - hmh;

	int tile_pos = threadIdx.x + threadIdx.y * blockDim.x;
	if(x_i < 0 || x_i >= w || y_i < 0 || y_i >= h){
			tile[tile_pos * 3] = tile[tile_pos * 3 + 1] = tile[tile_pos * 3 + 2] = 0;
	}else{
			int pos_i = x_i + y_i * w;
			tile[tile_pos * 3 + 0] = src[pos_i * 3];
			tile[tile_pos * 3 + 1] = src[pos_i * 3 + 1];
			tile[tile_pos * 3 + 2] = src[pos_i * 3 + 2];
	}

	__syncthreads();

	if(x_o >= w || y_o >= h)
			return;

	if(threadIdx.x >= TILE_SIZE_X || threadIdx.y >= TILE_SIZE_Y){
			return;
	}

	int tmp_x, tmp_y, tmp_pos, mask_pos;
	double tmp[] = {0, 0, 0};
	for(int i = 0; i < mh; i++){
			tmp_y = threadIdx.y + i;
			for(int j = 0; j < mw; j++){
					tmp_x = threadIdx.x + j;
					tmp_pos = tmp_x + tmp_y * blockDim.x;
					mask_pos = j + i * mw;
					tmp[0] += tile[tmp_pos * 3 + 0] * c_mask[mask_pos];
					tmp[1] += tile[tmp_pos * 3 + 1] * c_mask[mask_pos];
					tmp[2] += tile[tmp_pos * 3 + 2] * c_mask[mask_pos];
			}
	}
	dest[pos_o * 3] = (unsigned char) tmp[0]; 
	dest[pos_o * 3 + 1] = (unsigned char) tmp[1]; 
	dest[pos_o * 3 + 2] = (unsigned char) tmp[2]; 

	//Tile e indeksiran na nivo na block
	//Odma gi isfrlame site outputs shto se out of bounds na src    
	//
}

void gpu::conv_3ch_tiled(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const double *mask_t, int mw, int mh)
{
	size_t size = w * h * 3 * sizeof(unsigned char);

	unsigned char *src_d;
	unsigned char *dest_d;

	hipMalloc((void **)&src_d, size);
	hipMalloc((void **)&dest_d, size);

	hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mask), mask_t, mw * mh * sizeof(double));

	int NUM_OF_THREADS = 32;
	int TILE_SIZE_X = NUM_OF_THREADS - mw + 1;
	int TILE_SIZE_Y = NUM_OF_THREADS - mh + 1;
	dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
	//? Mozhe da se optimizira ova
	int GRID_SIZE_X = (int)ceil((float)w / TILE_SIZE_X);
	int GRID_SIZE_Y = (int)ceil((float)h / TILE_SIZE_Y);
	dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
	g_conv_3ch_tiled<<<gridSize, blockSize, blockSize.x * blockSize.y * sizeof(unsigned char) * 3>>>(src_d, dest_d, w, h, mw, mh, TILE_SIZE_X, TILE_SIZE_Y);

	hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(dest_d);
}

//-------------------1CHANNEL---------------------
__global__ void g_conv(const unsigned char* src, unsigned char* dest, int w, int h, const double* mask, int mw, int mh)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x >= w || y >= h)
		return;

	int pos = y * w + x;

	int hmw = mw >> 2;
	int hmh = mh >> 2;

	int start_x = x - hmw;
	int start_y = y - hmh;

	int tmp = 0;
	for(int i = start_y; i < start_y + mh; i++)
	{
		int tmp_y = start_y + i;
		if(tmp_y < 0 || tmp_y >= h)
			continue;
		for(int j = start_x; j < start_x + mw; j++)
		{
			int tmp_x = start_x + j;
			if(tmp_x < 0 || tmp_x >= h)
				continue;

			int src_pos = tmp_y * w + tmp_x;
			int mask_pos = i * w + j;
			tmp += src[src_pos] * mask[mask_pos];
		}
	}

	dest[pos] = tmp;
}

void gpu::conv(const unsigned char* src, unsigned char* dest, int w, int h, const double* mask, int mw, int mh)
{
	unsigned char* src_d;
	unsigned char* dest_d;
	unsigned char* mask_d;

	size_t size = w * h * sizeof(unsigned char);
	size_t mask_size = mw * mh * sizeof(double);

	hipMalloc((void**) &src_d, size);
	hipMalloc((void**) &mask_d, size);
	hipMalloc((void**) &dest_d, size);

	hipMemcpy(src_d, src, size, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, mask_size, hipMemcpyHostToDevice);

	int NUM_OFTHREADS = 32;
	dim3 BLOCK_SIZE(NUM_OFTHREADS, NUM_OFTHREADS); 
	int GRID_SIZE_X = (int) ceil((float) w / NUM_OFTHREADS);
	int GRID_SIZE_Y = (int) ceil((float) h / NUM_OFTHREADS);
	dim3 GRID_SIZE(GRID_SIZE_X, GRID_SIZE_Y); 

	g_conv<<<GRID_SIZE, BLOCK_SIZE>>>(src_d, dest_d, w, h, mask, mw, mh);

	hipMemcpy(dest, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(mask_d);
	hipFree(dest_d);
}

__global__ void g_conv_constant(const unsigned char* src, unsigned char* dest, int w, int h, int mw, int mh)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pos = y * w + x;

	int hmw = mw >> 2;
	int hmh = mh >> 2;

	int start_x = x - hmw;
	int start_y = y - hmh;

	unsigned char tmp = 0;
	for(int i = 0; i < mh; i++)
	{
		int tmp_y = start_y + i;
		if(tmp_y < 0 || tmp_y >= h)
			continue;
		for(int j = 0; j < mw; j++)
		{
			int tmp_x = start_x + j;
			if(tmp_x < 0 || tmp_x >= w)
				continue;

			int mask_pos = i * mw + j;
			int tmp_pos = tmp_y * w + tmp_x;
			tmp += c_mask[mask_pos] * src[tmp_pos];
		}
	}

	dest[pos] = tmp;
}

void gpu::conv_constant(const unsigned char* src_h, unsigned char* dest_h, int w, int h, const double* mask, int mw, int mh)
{
	unsigned char* src_d;
	unsigned char* dest_d;

	size_t size = w * h * sizeof(unsigned char);
	size_t mask_size = mw * mh * sizeof(double);

	hipMalloc((void**) &src_d, size);
	hipMalloc((void**) &dest_d, size);

	hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mask), mask, mask_size);

	int NUM_OF_THREADS = 32;
	dim3 BLOCK_SIZE(NUM_OF_THREADS, NUM_OF_THREADS);
	int GRID_SIZE_X = ceil(((double) w) / BLOCK_SIZE.x);
	int GRID_SIZE_Y = ceil(((double) h) / BLOCK_SIZE.y);
	dim3 GRID_SIZE(GRID_SIZE_X, GRID_SIZE_Y);
	g_conv_constant<<<GRID_SIZE, BLOCK_SIZE>>>(src_h, dest_h, w, h, mw, mh);

	hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(dest_d);
}

__global__ void g_conv_tiled(const unsigned char* src, unsigned char* dest, int w, int h, int mw, int mh) 
{
	extern __shared__ unsigned char tile[];

	int hmw = mw >> 2;
	int hmh = mh >> 2;
	//position in the shared memory tile
	int x_t = threadIdx.x + hmw;
	int y_t = threadIdx.y + hmh;

	//position that the tile has to load into memory
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//offsets of additional loads
	int x_offset = 0;
	int y_offset = 0;

	//top and bottom excess
	if(x_t / hmw == 0)
	{
		x_offset = - x_t % hmw;
	}
	else if((blockDim.x - x_t) / hmw == 0)
	{
		x_offset = (blockDim.x - x_t) % hmw;
	}

	//left and right excess
	if(y_t / hmh == 0)
	{
		y_offset = - y_t % hmh;
	}
	else if((blockDim.y - y_t) / hmh == 0)
	{
		y_offset = (blockDim.y - y_t) % hmh;
	}

	//corner offsets
	if(x_t / hmw == 1)
	{
		x_offset = - hmw - x_t % hmw;
	}
	else if((blockDim.x - x_t) / hmw == 1)
	{
		x_offset = hmw + (blockDim.x - x_t) % hmw;
	}
	if(y_t / hmh == 1)
	{
		y_offset = - hmh - y_t % hmh;
	}
	else if((blockDim.y - y_t) / hmh == 1)
	{
		y_offset = hmh + (blockDim.y - y_t) % hmh;
	}

	//load regular tile
	int tile_pos = y_t * blockDim.x + x_t;
	int src_pos  = y * w + x;
	tile[tile_pos] = src[src_pos];

	//deka ne se odnesuva na statichna memorija mozhe ova slobodno da ne se proveri
	int offset_tile_pos = (y_t + y_offset) * w + x_t + x_offset;

	int offset_x = x + x_offset;
	int offset_y = y + y_offset;
	int offset_src_pos = offset_y * w + offset_x;
	if(offset_x < 0 || offset_x >= w || offset_y < 0 || offset_y >= h)
		tile[offset_tile_pos] = 0;
	else
		tile[offset_tile_pos] = src[offset_src_pos];

	__syncthreads();
	
	int pos = src_pos;
//	int start_x = x_t - hmw;
//	int start_y = y_t - hmh;
	//isto e so gornoto
	int start_x = threadIdx.x;
	int start_y = threadIdx.y;
	unsigned char tmp = 0;

	//ne mora checks deka znam deka sekogash se validni poziciite vo tile
	for(int i = 0; i < mh; i++)
	{
		for(int j = 0; j < mw; j++)
		{
			int mask_pos = i * mw + j;
			int tmp_tile_pos = (start_y + i) * blockDim.x + start_x + j;
			tmp += c_mask[mask_pos] * tmp_tile_pos;
		}
	}

	dest[pos] = tmp;
}

//TODO: Ova ne e celosno testirano, bi trebalo da raboti ama ne sum siguren
void gpu::conv_tiled(const unsigned char* src_h, unsigned char* dest_h, int w, int h, const double* mask, int mw, int mh)
{
	unsigned char* src_d;
	unsigned char* dest_d;

	size_t size = w * h * sizeof(unsigned char);
	size_t mask_size = mw * mh * sizeof(double);

	hipMalloc((void**) &src_d, size);
	hipMalloc((void**) &dest_d, size);

	hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mask), mask, mask_size);

	int NUM_OF_THREADS = 32;
	dim3 BLOCK_SIZE(NUM_OF_THREADS, NUM_OF_THREADS);
	int GRID_SIZE_X = ceil(((double) w) / BLOCK_SIZE.x);
	int GRID_SIZE_Y = ceil(((double) h) / BLOCK_SIZE.y);
	dim3 GRID_SIZE(GRID_SIZE_X, GRID_SIZE_Y);
	g_conv_tiled<<<GRID_SIZE, BLOCK_SIZE>>>(src_h, dest_h, w, h, mw, mh);

	hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

	hipFree(src_d);
	hipFree(dest_d);
}
